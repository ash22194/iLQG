
#include <hip/hip_runtime.h>
__global__ void calc_average5(double* Xi1, const double* Xi2, const double* Xi3, const double* Xi4, const double* Xi5, const double* V,
    const int Nx1, const int Nx2, const int Nx3, const int Nx4, const int Nx5,
    const double dx1, const double dx2, const double dx3, const double dx4, const double dx5,
    const double minx1, const double minx2, const double minx3, const double minx4, const double minx5,  const int* corners_index)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x * blockDim.y
        + (threadIdx.x + threadIdx.y * blockDim.x);
    int num_threads = blockDim.x * blockDim.y * gridDim.x * gridDim.y;

    int grid_size = Nx1 * Nx2 * Nx3 * Nx4 * Nx5;
    int loc1, loc2, loc3, loc4, loc5, scalar_loc;
    double xi2, xi3, xi4, xi5;

    while (index < grid_size)
    {
        Xi1[index] = (Xi1[index] - minx1) / dx1;
        xi2 = (Xi2[index] - minx2) / dx2;
        xi3 = (Xi3[index] - minx3) / dx3;
        xi4 = (Xi4[index] - minx4) / dx4;
        xi5 = (Xi5[index] - minx5) / dx5;

        loc1 = min(Nx1 - 2, int(floor(Xi1[index])));
        loc2 = min(Nx2 - 2, int(floor(xi2)));
        loc3 = min(Nx3 - 2, int(floor(xi3)));
        loc4 = min(Nx4 - 2, int(floor(xi4)));
        loc5 = min(Nx5 - 2, int(floor(xi5)));

        Xi1[index] = Xi1[index] - loc1; // Weights from the 0 corner
        xi2 = xi2 - loc2;
        xi3 = xi3 - loc3;
        xi4 = xi4 - loc4;
        xi5 = xi5 - loc5;

        scalar_loc = loc1 + Nx1 * (loc2 + Nx2 * (loc3 + Nx3 * (loc4 + Nx4 * loc5)));

        Xi1[index] = (1 - Xi1[index]) * ((1 - xi2) * ((1 - xi3) * ((1 - xi4) * ((1 - xi5) * V[scalar_loc + corners_index[0]]
                                                                                    + xi5 * V[scalar_loc + corners_index[1]])
                                                                       + xi4 * ((1 - xi5) * V[scalar_loc + corners_index[2]]
                                                                                    + xi5 * V[scalar_loc + corners_index[3]]))
                                                          + xi3 * ((1 - xi4) * ((1 - xi5) * V[scalar_loc + corners_index[4]]
                                                                                    + xi5 * V[scalar_loc + corners_index[5]])
                                                                       + xi4 * ((1 - xi5) * V[scalar_loc + corners_index[6]]
                                                                                    + xi5 * V[scalar_loc + corners_index[7]])))
                                             + xi2 * ((1 - xi3) * ((1 - xi4) * ((1 - xi5) * V[scalar_loc + corners_index[8]]
                                                                                    + xi5 * V[scalar_loc + corners_index[9]])
                                                                       + xi4 * ((1 - xi5) * V[scalar_loc + corners_index[10]]
                                                                                    + xi5 * V[scalar_loc + corners_index[11]]))
                                                          + xi3 * ((1 - xi4) * ((1 - xi5) * V[scalar_loc + corners_index[12]]
                                                                                    + xi5 * V[scalar_loc + corners_index[13]])
                                                                       + xi4 * ((1 - xi5) * V[scalar_loc + corners_index[14]]
                                                                                    + xi5 * V[scalar_loc + corners_index[15]]))))
                         + Xi1[index] * ((1 - xi2) * ((1 - xi3) * ((1 - xi4) * ((1 - xi5) * V[scalar_loc + corners_index[16]]
                                                                                    + xi5 * V[scalar_loc + corners_index[17]])
                                                                       + xi4 * ((1 - xi5) * V[scalar_loc + corners_index[18]]
                                                                                    + xi5 * V[scalar_loc + corners_index[19]]))
                                                          + xi3 * ((1 - xi4) * ((1 - xi5) * V[scalar_loc + corners_index[20]]
                                                                                    + xi5 * V[scalar_loc + corners_index[21]])
                                                                       + xi4 * ((1 - xi5) * V[scalar_loc + corners_index[22]]
                                                                                    + xi5 * V[scalar_loc + corners_index[23]])))
                                             + xi2 * ((1 - xi3) * ((1 - xi4) * ((1 - xi5) * V[scalar_loc + corners_index[24]]
                                                                                    + xi5 * V[scalar_loc + corners_index[25]])
                                                                       + xi4 * ((1 - xi5) * V[scalar_loc + corners_index[26]]
                                                                                    + xi5 * V[scalar_loc + corners_index[27]]))
                                                          + xi3 * ((1 - xi4) * ((1 - xi5) * V[scalar_loc + corners_index[28]]
                                                                                    + xi5 * V[scalar_loc + corners_index[29]])
                                                                       + xi4 * ((1 - xi5) * V[scalar_loc + corners_index[30]]
                                                                                    + xi5 * V[scalar_loc + corners_index[31]]))));
        index = index + num_threads;
    }
}