
#include <hip/hip_runtime.h>
__global__ void calc_average2(double * Xi1, const double * Xi2, const double * V, const int query_grid_size, const int Nx1, const int Nx2, const double dx1, const double dx2, const double minx1, const double minx2, const int * corners_index)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x * blockDim.y
                + (threadIdx.x + threadIdx.y * blockDim.x);
    int num_threads = blockDim.x * blockDim.y * gridDim.x * gridDim.y;
//     int query_grid_size = Nx1 * Nx2;
    int loc1, loc2, scalar_loc;
    double xi2;

    while (index < query_grid_size)
    {
        Xi1[index] = (Xi1[index] - minx1) / dx1;
        xi2        = (Xi2[index] - minx2) / dx2;
        
        loc1 = min(Nx1 - 2, int(floor(Xi1[index])));
        loc2 = min(Nx2 - 2, int(floor(xi2)));
        
        Xi1[index] = Xi1[index] - loc1; // Weights from the 0 corner
        xi2        = xi2 - loc2;
        
        scalar_loc = loc1 + Nx1 * loc2;
        
        Xi1[index] = (1 - Xi1[index]) * ((1 - xi2) * V[scalar_loc + corners_index[0]]
                                             + xi2 * V[scalar_loc + corners_index[1]])
                         + Xi1[index] * ((1 - xi2) * V[scalar_loc + corners_index[2]]
                                             + xi2 * V[scalar_loc + corners_index[3]]);
        index = index + num_threads;
    }
        
}