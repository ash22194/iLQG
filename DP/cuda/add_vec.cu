
#include <hip/hip_runtime.h>
__global__ void add_vec( double * v1, const double * v2 ) 
{
    int idx = threadIdx.x;
    v1[idx] += v2[idx];
}