
#include <hip/hip_runtime.h>
__global__ void calc_average2(double * Xi1, const double * Xi2, const double * V, const int Nx1, const int Nx2, const double dx1, const double dx2, const double minx1, const double minx2, const int * corners_index)
{
    int index = blockIdx.x + blockIdx.y * gridDim.x 
                + (threadIdx.x + threadIdx.y * blockDim.x);
    int num_threads = blockDim.x * blockDim.y * gridDim.x * gridDim.y;
    
    while (index < Nx1)
    {
        Xi1[index] = (Xi1[index] - minx1) / dx1;
        double xi2 = (Xi2[index] - minx2) / dx2;
        
        int loc1 = min(Nx1 - 2, int(floor(Xi1[index])));
        int loc2 = min(Nx2 - 2, int(floor(xi2)));
        
        Xi1[index] = Xi1[index] - loc1; // Weights from the 0 corner
        xi2        = xi2 - loc2;
        
        int scalar_loc = loc1 + Nx1 * loc2;
        
        Xi1[index] = (1 - Xi1[index]) * ((1 - xi2) * V[scalar_loc + corners_index[0]]
                                             + xi2 * V[scalar_loc + corners_index[1]])
                         + Xi1[index] * ((1 - xi2) * V[scalar_loc + corners_index[2]]
                                             + xi2 * V[scalar_loc + corners_index[3]]);
        index = index + num_threads;
    }
        
}